
#include <hip/hip_runtime.h>
template <typename T>  __global__  void kernelgpuInitq(T *f, T *xdg, T *uinf, T *param, int modelnumber, int ng, int ncx, int nce, int npe, int ne)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	while (i<ng) {
		int j = i%npe;
		int k = (i-j)/npe;
		T xdg1 = xdg[j+npe*0+npe*ncx*k];
		T xdg2 = xdg[j+npe*1+npe*ncx*k];
		f[j+npe*0+npe*nce*k] = sin(xdg1*3.141592653589793)*sin(xdg2*3.141592653589793);
		f[j+npe*1+npe*nce*k] = 0.0;
		f[j+npe*2+npe*nce*k] = 0.0;
		i += blockDim.x * gridDim.x;
	}
}

template <typename T> void gpuInitq(T *f, T *xdg, T *uinf, T *param, int modelnumber, int ng, int ncx, int nce, int npe, int ne)
{
	int blockDim = 256;
	int gridDim = (ng + blockDim - 1) / blockDim;
	gridDim = (gridDim>1024)? 1024 : gridDim;
	kernelgpuInitq<<<gridDim, blockDim>>>(f, xdg, uinf, param, modelnumber, ng, ncx, nce, npe, ne);
}

template void gpuInitq(double *, double *, double *, double *, int, int, int, int, int, int);
template void gpuInitq(float *, float *, float *, float *, int, int, int, int, int, int);
